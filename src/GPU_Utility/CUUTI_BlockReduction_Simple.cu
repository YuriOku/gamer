#include "hip/hip_runtime.h"
#include "Macro.h"

#ifdef GPU


// check
// one must define RED_NTHREAD for the reduction kernel in advance since we use the static shared memory
#ifndef RED_NTHREAD
#  error : ERROR : RED_NTHREAD is not defined in BlockReduction_Simple !!
#endif


// define the reduction operation here
#if   defined RED_SUM
#  define RED( a, b )   ( (a) + (b) )
#elif defined RED_MAX
#  define RED( a, b )   MAX( (a), (b) )
#elif defined RED_MIN
#  define RED( a, b )   MIN( (a), (b) )
#else
#  error : undefined reduction operation !!
#endif




//-------------------------------------------------------------------------------------------------------
// Function    :  BlockReduction_Simple
// Description :  GPU reduction within each thread block using the explicit synchronization
//
// Note        :  1. Mainly used for the DCUs
//                   --> BlockReduction_WarpSync() and BlockReduction_Shuffle() fail on the DCUs
//                2. Must define RED_NTHREAD in advance since we use the static shared memory
//                   --> RED_NTHREAD must < 2048
//                3. Must define either RED_SUM, RED_MAX, or RED_MIN in advance to determine the reduction operation
//                4. Only thread 0 will hold the correct result after calling this function
//
// Parameter   :  val : Per-thread value for the reduction
//
// Return value:  Reduction of "val"
//---------------------------------------------------------------------------------------------------
__inline__ __device__
real BlockReduction_Simple( real val )
{

   const uint tid_x     = threadIdx.x;
   const uint tid_y     = threadIdx.y;
   const uint tid_z     = threadIdx.z;
   const uint bdim_x    = blockDim.x;
   const uint bdim_y    = blockDim.y;
   const uint ID        = __umul24( tid_z, __umul24(bdim_x,bdim_y) ) + __umul24( tid_y, bdim_x ) + tid_x;
   const uint FloorPow2 = 1 << ( 31-__clz(RED_NTHREAD) );   // largest power-of-two value not greater than RED_NTHREAD
   const uint Remain    = RED_NTHREAD - FloorPow2;

   __shared__ real s_Reduction[RED_NTHREAD];


// store values for the reduction to the shared memory
   s_Reduction[ID] = val;
   __syncthreads();


// perform reduction for the elements larger than FloorPow2 to ensure that the number of remaining elements is power-of-two
   if ( ID < Remain )   s_Reduction[ID] = RED( s_Reduction[ID], s_Reduction[ ID + FloorPow2 ] );
   __syncthreads();


// parallel reduction with the shared memory
#  if ( RED_NTHREAD >= 2048 )
#  error : ERROR : RED_NTHREAD >= 2048 !!
#  endif

#  if ( RED_NTHREAD >= 1024 )
   if ( ID < 512 )   s_Reduction[ID] = RED( s_Reduction[ID], s_Reduction[ ID + 512 ] );   __syncthreads();
#  endif

#  if ( RED_NTHREAD >= 512 )
   if ( ID < 256 )   s_Reduction[ID] = RED( s_Reduction[ID], s_Reduction[ ID + 256 ] );   __syncthreads();
#  endif

#  if ( RED_NTHREAD >= 256 )
   if ( ID < 128 )   s_Reduction[ID] = RED( s_Reduction[ID], s_Reduction[ ID + 128 ] );   __syncthreads();
#  endif

#  if ( RED_NTHREAD >= 128 )
   if ( ID <  64 )   s_Reduction[ID] = RED( s_Reduction[ID], s_Reduction[ ID +  64 ] );   __syncthreads();
#  endif

#  if ( RED_NTHREAD >= 64 )
   if ( ID <  32 )   s_Reduction[ID] = RED( s_Reduction[ID], s_Reduction[ ID +  32 ] );   __syncthreads();
#  endif

#  if ( RED_NTHREAD >= 32 )
   if ( ID <  16 )   s_Reduction[ID] = RED( s_Reduction[ID], s_Reduction[ ID +  16 ] );   __syncthreads();
#  endif

#  if ( RED_NTHREAD >= 16 )
   if ( ID <   8 )   s_Reduction[ID] = RED( s_Reduction[ID], s_Reduction[ ID +   8 ] );   __syncthreads();
#  endif

#  if ( RED_NTHREAD >= 8 )
   if ( ID <   4 )   s_Reduction[ID] = RED( s_Reduction[ID], s_Reduction[ ID +   4 ] );   __syncthreads();
#  endif

#  if ( RED_NTHREAD >= 4 )
   if ( ID <   2 )   s_Reduction[ID] = RED( s_Reduction[ID], s_Reduction[ ID +   2 ] );   __syncthreads();
#  endif

#  if ( RED_NTHREAD >= 2 )
   if ( ID <   1 )   s_Reduction[ID] = RED( s_Reduction[ID], s_Reduction[ ID +   1 ] );   __syncthreads();
#  endif

   return s_Reduction[0];

} // FUNCTION : BlockReduction_Simple

#undef RED


#endif // #ifdef GPU
